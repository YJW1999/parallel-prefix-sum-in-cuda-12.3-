﻿#pragma once

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"

#include <iostream>

#define size 500

__device__ int ScanBlock_HillisSteele(int val, int* smem)
{
	smem[threadIdx.x] = val;
	__syncthreads();

	for (int offset = 1; offset < blockDim.x; offset <<= 1)
	{
		if (threadIdx.x >= offset) val += smem[threadIdx.x-offset];
		__syncthreads();

		smem[threadIdx.x] = val;
		__syncthreads();
	}

	return smem[threadIdx.x];
}

__device__ int DeviceScan(int val, int* smem, int* sum, int* counter) {
	val = ScanBlock_HillisSteele(val, smem);
	__shared__ int offset;
	if (threadIdx.x == blockDim.x - 1) {
		while (atomicAdd(counter, 0) < blockIdx.x);
			
		offset = atomicAdd(sum, val);
		// Use memory fence to ensure the order of atomics
		__threadfence();
		// Signalize the next block that can be processed
		atomicAdd(counter, 1);
	}

	__syncthreads();

	return offset + val;
}

__global__ void parallelPrefixKernel(const int* input, int* output, int _size, int* sum, int* counter) {
	extern __shared__ int smem[];
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + tid;

	int val = 0;
	if (gid < _size) val = input[gid];

	val = DeviceScan(val, smem, sum, counter);

	if (gid < _size) {
		output[gid] = val;
	}
}
 
int main() {
	int* data = new int[size];
	for (int i = 0; i < size; ++i) {
		data[i] = rand() % 1000;
	}

	int* input_data, * output_data;
	hipMalloc((void**)&input_data, size * sizeof(int));
	hipMalloc((void**)&output_data, size * sizeof(int));
	hipMemcpy(input_data, data, size * sizeof(int), hipMemcpyHostToDevice);

	int* host_sum = 0, * host_counter = 0;

	int* sum, *counter;
	hipMalloc((void**)&sum, sizeof(int));
	hipMalloc((void**)&counter, sizeof(int));
	hipMemcpy(sum, host_sum, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(counter, host_counter, sizeof(int), hipMemcpyHostToDevice);
	
	//set configuration for block size and number of blocks
	const int blockSize = 256;
	const int gridSize = (size + blockSize - 1) / blockSize;

	//cuda Kernel function call
	parallelPrefixKernel <<<gridSize, blockSize >>> (input_data, output_data, size, sum, counter);

	int* output = new int[size];
	hipMemcpy(output, output_data, size * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < size; ++i) {
		std::cout << data[i] << ": " << output[i] << std::endl;
	}

	return 0;
}
